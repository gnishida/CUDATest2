#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "simple_derivation.cuh"
#include "lsystem.cuh"

const int NUM_THREADS = 16;
const int N = 10;

__global__ 
void hello(float *result)  {
	int id = threadIdx.x;// + blockIdx.x * 64;
	hiprandState state;
	hiprand_init(1234, id, 0, &state);

	Literal l(0);

	/*String model = Literal("X", 0, 124.0f, 0.0f);
	vector<Action> actions = getActions(model);
	int index = hiprand_uniform(&state) * actions.size();
	model = actions[index].apply(model);*/


}
 
void cudaMain() {
	int size = sizeof(float) * NUM_THREADS * N;

	float* hResult = (float*)malloc(size);
	float* dResult;

	hipMalloc((void**)&dResult, size); 
	hipMemcpy(dResult, hResult, size, hipMemcpyHostToDevice);
	
	hello<<<1, NUM_THREADS>>>(dResult);
	hipMemcpy(hResult, dResult, size, hipMemcpyDeviceToHost);
	hipFree(dResult);

	for (int i = 0; i < NUM_THREADS; ++i) {
		for (int j = 0; j < N; ++j) {
			printf("%lf, ", hResult[i * N + j]);
		}
		printf("\n");
	}
}