#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "pinned_memory_test.cuh"

using namespace std;

const int NUM_THREADS = 256;

__global__ 
void mattest(float* matA, float* matB, unsigned char* mask, int* size, float *result)  {
	int id = threadIdx.x + blockIdx.x * NUM_THREADS;

	__shared__ float count;
	__shared__ float total;
	
	if (id == 0) {
		count = 0.0;
		total = 0.0;
	}
	int s = size[0];

	__syncthreads();

	int iterations = ceil((float)s / NUM_THREADS);

	for (int iter = 0; iter < iterations; ++iter) {
		int index = iter * NUM_THREADS + id;
		if (index < s) {
			if (mask[index] > 0) {
				float a = matA[index];
				float b = matB[index];

				if (a > 0.5) {
					if (b < 0.5) {
						atomicAdd(&count, 1);
					}
					atomicAdd(&total, 1);
				} else {
					if (b > 0.5) {
						atomicAdd(&count, 1);
					}
				}
			}
		}
	}

	__syncthreads();

	if (id == 0) {
		result[0] = count;
		result[1] = total;
	}
}
 
/**
 * 2つの行列の差分の数を計算する。
 */
void cudaMain(cv::Mat_<float>& a, cv::Mat_<float>& b, cv::Mat_<uchar>& mask) {
	// ホスト側でメモリ確保
	float* hMatA;// = (float*)malloc(sizeof(float) * a.rows * a.cols);
	hipHostAlloc((void**)&hMatA, sizeof(float) * a.rows * a.cols, hipHostMallocDefault);
	float* hMatB;// = (float*)malloc(sizeof(float) * b.rows * b.cols);
	hipHostAlloc((void**)&hMatB, sizeof(float) * b.rows * b.cols, hipHostMallocDefault);
	unsigned char* hMask;// = (unsigned char*)malloc(sizeof(unsigned char) * mask.rows * mask.cols);
	hipHostAlloc((void**)&hMask, sizeof(unsigned char) * mask.rows * mask.cols, hipHostMallocDefault);
	//int hSize = a.rows * a.cols;
	int* hSize;
	hipHostAlloc((void**)&hSize, sizeof(int), hipHostMallocDefault);
	hSize[0] = a.rows * a.cols;
	float* hResult;// = (float*)malloc(sizeof(float) * 2);
	hipHostAlloc((void**)&hResult, sizeof(float) * 2, hipHostMallocDefault);

	// デバイス側の変数
	float* dMatA;
	float* dMatB;
	unsigned char* dMask;
	int* dSize;
	float* dResult;

	memcpy(hMatA, a.data, sizeof(float) * a.rows * a.cols);
	memcpy(hMatB, b.data, sizeof(float) * b.rows * b.cols);
	memcpy(hMask, mask.data, sizeof(unsigned char) * mask.rows * mask.cols);

	hipMalloc((void**)&dMatA, sizeof(float) * a.rows * a.cols); 
	hipMemcpy(dMatA, hMatA, sizeof(float) * a.rows * a.cols, hipMemcpyHostToDevice);

	hipMalloc((void**)&dMatB, sizeof(float) * b.rows * b.cols); 
	hipMemcpy(dMatB, hMatB, sizeof(float) * b.rows * b.cols, hipMemcpyHostToDevice);

	hipMalloc((void**)&dMask, sizeof(unsigned char) * mask.rows * mask.cols); 
	hipMemcpy(dMask, hMask, sizeof(unsigned char) * mask.rows * mask.cols, hipMemcpyHostToDevice);

	hipMalloc((void**)&dSize, sizeof(int)); 
	hipMemcpy(dSize, hSize, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&dResult, sizeof(float) * 2);

	mattest<<<1, NUM_THREADS>>>(dMatA, dMatB, dMask, dSize, dResult);

	hipMemcpy(hResult, dResult, sizeof(float) * 2, hipMemcpyDeviceToHost);

	hipFree(dMatA);
	hipFree(dMatB);
	hipFree(dMask);
	hipFree(dSize);
	hipFree(dResult);

	//cout << "Result: " << hResult[0] << ", " << hResult[1] << endl;
	//cout << "Time2: " << t1 << "," << t2 << "," << t3 << "," << t4 << "," << t5 << endl;

	//free(hMatA);
	//free(hMatB);
	//free(hMask);
	//free(hSize);
	//free(hResult);

	hipHostFree(hMatA);
	hipHostFree(hMatB);
	hipHostFree(hMask);
	hipHostFree(hSize);
	hipHostFree(hResult);
}